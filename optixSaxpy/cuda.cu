#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>

__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

void saxpy_optix(int N, float a, float *dx, float *dy);

int main(void) {
  int N = 1 << 29;
  float *x, *y, *d_x, *d_y;
  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  printf("Using optix kernels\n");
  saxpy_optix(N, 2.0f, d_x, d_y);

  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
  printf("Using cuda kernels\n");
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i] - 4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}
